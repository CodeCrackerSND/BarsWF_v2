#include "hip/hip_runtime.h"
#include <stdio.h>
//we obviously using CUDA :-D
#define CUDA

//#define DebugValues  // if debug values print values!

#include <hip/hip_runtime.h>
#include "md5_kernel.h"
#include "md5.h"
#include <stdio.h>
#include "cuPrintf.cu"

//__constant__
//__device__ __constant__ unsigned int charset_c[256];// = {0xb182b498, 0xf4d2ac41, 0x1f636569, 0xaf4caf00};
//__device__ __constant__ unsigned int target_hash[4];// = {0xb182b498, 0xf4d2ac41, 0x1f636569, 0xaf4caf00};

 __shared__ unsigned int charset_c[256];
 __shared__ unsigned int target_hash[4];

void CCU2()
{
	hipError_t result = hipGetLastError();
	if(result != hipSuccess)
	{
		char msg[512];msg[0]=0;
		strcpy_s(msg, 512, "CUDA kernel error: ");
		strcat_s(msg, 512, hipGetErrorString(result));
		printf(msg);
		return;
	}
}

extern "C" void cuda_get_mem(gpu_data_xyz* data)
{
	//get memory for data transfers

	// Alocate only once
	//hipMalloc( (void**)&charset_c, 256*sizeof(unsigned int));
	//hipMalloc( (void**)&target_hash, 4*sizeof(unsigned int));
	//CCU2();

//	hipMalloc    ((void **) &target_hash , sizeof(int)*4);
	//CCU2();
	hipMalloc    ((void **) &data->data_d      , sizeof(int)*4*data->thread_n*data->grid_n);
	CCU2();
	hipHostMalloc((void **) &data->data_h, sizeof(int)*4*data->thread_n*data->grid_n);
	CCU2();

	hipMalloc    ((void **) &data->result_d    , sizeof(int)*data->thread_n*data->grid_n);   
	CCU2();
	hipHostMalloc((void **) &data->result, sizeof(int)*data->thread_n*data->grid_n);
	CCU2();
}

__global__ void initShared(int* hash_i, unsigned char *charset, int charset_len)
{

    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // load to shared memory array
    // assumes Mystruct has correct copy assignment semantics
    //d_s[threadIdx.x] = theStructArray[tid]

	for(int i=0;i<charset_len;i++)
		charset_c[threadIdx.x+i] = charset[tid+i];

	for(int i=0;i<4;i++)
		target_hash[threadIdx.x+i] = hash_i[tid+i];

    __syncthreads();

    // Each thread has now loaded one value to the block scoped shared value
}


extern "C" void init_md5_cuda(gpu_data_xyz* data, int* hash_i, unsigned char *charset, int charset_len)
{

	initShared<<<data->grid_n, 1>>>(hash_i, charset, charset_len);

    //hipMemcpy( cuda_x, &x, sizeof( float ), hipMemcpyHostToDevice );

	//int charset_int[256];
	//memset(charset_int, 0, sizeof(charset_int));
	//for(int i=0;i<charset_len;i++)
	//	charset_int[i] = charset[i];
/*
	for(int i=0;i<4;i++)
		charset_int[100+i] = hash_i[i];*/
	
	memset(data->result, 0, sizeof(int)*data->thread_n*data->grid_n);
	hipMemcpy(data->result_d, data->result, sizeof(int)*data->thread_n*data->grid_n, hipMemcpyHostToDevice);
	CCU2();
	
	//hipMemcpy( charset_c, &charset_int, sizeof(unsigned int)*charset_len, hipMemcpyHostToDevice );

	//hipMemcpyToSymbol(HIP_SYMBOL(charset_c), &charset_int, sizeof(unsigned int)*charset_len);
	//hipMemcpyToSymbol(HIP_SYMBOL("charset_c"), charset_int, sizeof(int)*charset_len);//
	//CCU2();

	//hipMemcpyToSymbol(HIP_SYMBOL(target_hash), &hash_i, sizeof(int)*4);
	//hipMemcpy(target_hash, hash_i, sizeof(unsigned int)*4, hipMemcpyHostToDevice);
	//CCU2();
	//	hipMemcpyToSymbol(HIP_SYMBOL(md5_const), md5_const_host, sizeof(md5_const));
}

extern "C" void cuda_free_mem(gpu_data_xyz* data)
{
	//hipFree(target_hash);  
	hipFree(data->data_d);  
	hipFree(data->result_d);  

	hipHostFree(data->data_h); 
	hipHostFree(data->result); 
}

__global__ void md5_gpu_bruteforce_thread(unsigned int *data_d, unsigned int *result_d, unsigned int pwd_len, int charset_len)
{

	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
	//unsigned int data[4];
	unsigned int data0, data1, data2, data3;
	int a,b,c,d;
	const int len = pwd_len*8;//code_len[0];//
	int ta,tb,tc,td;

	data0=data_d[ix*4+0];
	data1=data_d[ix*4+1];
	data2=data_d[ix*4+2];
	data3=data_d[ix*4+3];

	//for(int i=0;i<4;i++)data[i]=data_d[ix*4+i];//
	//		if(s3_limit==26)data[0]=(data[0]&0xff00ffff)+(('a'+s3)<<16);//only if 3 symbols to brute force
		
	int res = KEY_NOT_FOUND;
	ta = target_hash[0];
	tb = target_hash[1];
	tc = target_hash[2];
	td = target_hash[3];



	// Unroll known steps:
	unrollII (tc, td, ta, tb, data2, S43, 0x2ad7d2bb); /* 63 */
	unrollII (td, ta, tb, tc, 0, S42, 0xbd3af235); /* 62 */
	unrollII (ta, tb, tc, td, 0, S41, 0xf7537e82); /* 61 */
	unrollII (tb, tc, td, ta, 0, S44, 0x4e0811a1); /* 60 */
	unrollII (tc, td, ta, tb, 0, S43, 0xa3014314); /* 59 */
	unrollII (td, ta, tb, tc, 0, S42, 0xfe2ce6e0); /* 58 */
	unrollII (ta, tb, tc, td, 0, S41, 0x6fa87e4f); /* 57 */
	unrollII (tb, tc, td, ta, data1, S44, 0x85845dd1); /* 56 */
	unrollII (tc, td, ta, tb, 0, S43, 0xffeff47d); /* 55 */
	unrollII (td, ta, tb, tc, data3, S42, 0x8f0ccc92); /* 54 */
	unrollII (ta, tb, tc, td, 0, S41, 0x655b59c3); /* 53 */
	unrollII (tb, tc, td, ta, 0, S44, 0xfc93a039); /* 52 */
	unrollII (tc, td, ta, tb, len, S43, 0xab9423a7); /* 51 */
	unrollII (td, ta, tb, tc, 0, S42, 0x432aff97); /* 50 */

	for(int s2=0;s2<charset_len;s2++)
	{
		data0=(data0&0xffff00ff)+((charset_c[s2])<<8);

		for(int s1=0;s1<charset_len;s1++)
		{
			data0=(data0&0xffffff00)+(charset_c[s1]);

			a=0x67452301;b=0xefcdab89;c=0x98badcfe;d=0x10325476;

			/* Round 1 */

			FF2_first (a, b, c, d, data0, S11, 0xd76aa478,0,0); /* 1 */
			FF2 (d, a, b, c, data1, S12, 0xe8c7b756,1,1); /* 2 */
			FF2 (c, d, a, b, data2, S13, 0x242070db,2,2); /* 3 */
			FF2 (b, c, d, a, data3, S14, 0xc1bdceee,3,3); /* 4 */


			FF2_NoData (a, b, c, d, 0, S11, 0xf57c0faf,4,15); /* 5 */ //5th and so forth is 0
			FF2_NoData (d, a, b, c, 0, S12, 0x4787c62a,5,15); /* 6 */
			FF2_NoData (c, d, a, b, 0, S13, 0xa8304613,6,15); /* 7 */
			FF2_NoData (b, c, d, a, 0, S14, 0xfd469501,7,15); /* 8 */
			FF2_NoData (a, b, c, d, 0, S11, 0x698098d8,8,15); /* 9 */
			FF2_NoData (d, a, b, c, 0, S12, 0x8b44f7af,9,15); /* 10 */
			FF2_NoData (c, d, a, b, 0, S13, 0xffff5bb1,10,15); /* 11 */
			FF2_NoData (b, c, d, a, 0, S14, 0x895cd7be,11,15); /* 12 */
			FF2_NoData (a, b, c, d, 0, S11, 0x6b901122,12,15); /* 13 */
			FF2_NoData (d, a, b, c, 0, S12, 0xfd987193,13,15); /* 14 */
			FF2 (c, d, a, b, len, S13, 0xa679438e,14,14); /* 15 */
			FF2_NoData (b, c, d, a, 0, S14, 0x49b40821,15,15); /* 16 */

			/* Round 2 */
			GG2 (a, b, c, d, data1, S21, 0xf61e2562,16,1); /* 17 */
			GG2_NoData (d, a, b, c, 0, S22, 0xc040b340,17,15); /* 18 */
			GG2_NoData (c, d, a, b, 0, S23, 0x265e5a51,18,15); /* 19 */
			GG2 (b, c, d, a, data0, S24, 0xe9b6c7aa,19,0); /* 20 */
			GG2_NoData (a, b, c, d, 0, S21, 0xd62f105d,20,15); /* 21 */
			GG2_NoData (d, a, b, c, 0, S22,  0x2441453,21,15); /* 22 */
			GG2_NoData (c, d, a, b, 0, S23, 0xd8a1e681,22,15); /* 23 */
			GG2_NoData (b, c, d, a, 0, S24, 0xe7d3fbc8,23,15); /* 24 */
			GG2_NoData (a, b, c, d, 0, S21, 0x21e1cde6,24,15); /* 25 */
			GG2 (d, a, b, c, len, S22, 0xc33707d6,25,14); /* 26 */
			GG2 (c, d, a, b, data3, S23, 0xf4d50d87,26,3); /* 27 */
			GG2_NoData (b, c, d, a, 0, S24, 0x455a14ed,27,15); /* 28 */
			GG2_NoData (a, b, c, d, 0, S21, 0xa9e3e905,28,15); /* 29 */
			GG2 (d, a, b, c, data2, S22, 0xfcefa3f8,29,2); /* 30 */
			GG2_NoData (c, d, a, b, 0, S23, 0x676f02d9,30,15); /* 31 */
			GG2_NoData (b, c, d, a, 0, S24, 0x8d2a4c8a,31,15); /* 32 */

			/* Round 3 */
			HH2_NoData (a, b, c, d, 0, S31, 0xfffa3942,32,15); /* 33 */
			HH2_NoData (d, a, b, c, 0, S32, 0x8771f681,33,15); /* 34 */
			HH2_NoData (c, d, a, b, 0, S33, 0x6d9d6122,34,15); /* 35 */
			HH2 (b, c, d, a, len, S34, 0xfde5380c,35,14); /* 36 */
			HH2 (a, b, c, d, data1, S31, 0xa4beea44,36,1); /* 37 */
			HH2_NoData (d, a, b, c, 0, S32, 0x4bdecfa9,37,15); /* 38 */
			HH2_NoData (c, d, a, b, 0, S33, 0xf6bb4b60,38,15); /* 39 */
			HH2_NoData (b, c, d, a, 0, S34, 0xbebfbc70,39,15); /* 40 */
			HH2_NoData (a, b, c, d, 0, S31, 0x289b7ec6,40,15); /* 41 */
			HH2 (d, a, b, c, data0, S32, 0xeaa127fa,41,0); /* 42 */
			HH2 (c, d, a, b, data3, S33, 0xd4ef3085,42,3); /* 43 */
			HH2_NoData (b, c, d, a, 0, S34,  0x4881d05,43,15); /* 44 */
			HH2_NoData (a, b, c, d, 0, S31, 0xd9d4d039,44,15); /* 45 */
			HH2_NoData (d, a, b, c, 0, S32, 0xe6db99e5,45,15); /* 46 */
			if(d!=td)continue;
			HH2_NoData (c, d, a, b, 0, S33, 0x1fa27cf8,46,15); /* 47 */
			HH2 (b, c, d, a, data2, S34, 0xc4ac5665,47,2); /* 48 */

			/* Round 4 */
			II2 (a, b, c, d, data0, S41, 0xf4292244,48,0); /* 49 */

			if(a==ta && b==tb && c==tc)
			{
				res = KEY_FOUND;

				#ifdef DebugValues  // finnaly print values
				//cuPrintf("a = %X, b = %X, c = %X, d = %X\n", a, b, c, d);
				//cuPrintf("ta = %X, tb = %X, tc = %X, td = %X\n", ta, tb, tc, td);

				//cuPrintf("th0 = %X, th1 = %X, th2 = %X, th3 = %X\n", target_hash[0], target_hash[1], target_hash[2], target_hash[3]);
	
	// For value unrolling:	
	cuPrintf("ta = %X, tb = %X, tc = %X, td = %X\n", ta_old, tb_old, tc_old, td_old);

				#endif

			};
		}
	}

	result_d[ix] = res;
}

// The host CPU Mandebrot thread spawner
// 0: OK
// 1: Error while calculation
extern "C" int do_123(gpu_data_xyz* data, int pwd_len, int charset_len)
{
	dim3 threads(data->thread_n);
	dim3 grid(data->grid_n);

	//copy combinations to device
/*	hipEvent_t stop;
	CUDA_SAFE_CALL( hipEventCreate(&stop)  );
	CCU2();*/

//	hipMemcpyAsync(data_d, data->data_h, sizeof(int)*4*data->thread_n*data->grid_n, hipMemcpyHostToDevice,0);  
	hipMemcpy(data->data_d, data->data_h, sizeof(int)*4*data->thread_n*data->grid_n, hipMemcpyHostToDevice);  
	CCU2();

#ifdef DebugValues
cudaPrintfInit();  // init print - first step of cuPrintf
#endif

	
	md5_gpu_bruteforce_thread<<<grid, threads>>>(data->data_d, data->result_d, pwd_len, charset_len);
	CCU2();

//	hipMemcpyAsync(data->result, result_d, sizeof(int)*data->thread_n*data->grid_n, hipMemcpyDeviceToHost, NULL);  
	hipMemcpy(data->result, data->result_d, sizeof(int)*data->thread_n*data->grid_n, hipMemcpyDeviceToHost);  
	CCU2();

#ifdef DebugValues
    cudaPrintfDisplay(stdout, true);  // part two of cuPrintf
    cudaPrintfEnd();
#endif

	hipError_t result = hipGetLastError();
	if(result != hipSuccess)
	{
		char msg[512];msg[0]=0;
		strcpy_s(msg, 512, "CUDA kernel error: ");
		strcat_s(msg, 512, hipGetErrorString(result));
		printf(msg);
		return 1;
	}

/*	hipEventRecord(stop, 0);
	while( hipEventQuery(stop) == hipErrorNotReady )
	{
		Sleep(3);
	}
	CUDA_SAFE_CALL( hipEventDestroy(stop));*/

	return 0;
}
